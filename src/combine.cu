#include <hip/hip_runtime.h>
#include <assert.h>
#include <iostream>
#include <sstream>
#include <fstream>

#define MAX_DIMS 10
#define TILE 32
#define BASE_THREAD_NUM 32

#define ADD_FUNC       1
#define MUL_FUNC       2
#define ID_FUNC        3
#define NEG_FUNC       4
#define LT_FUNC        5
#define EQ_FUNC        6
#define SIGMOID_FUNC   7
#define RELU_FUNC      8
#define RELU_BACK_FUNC 9
#define LOG_FUNC       10
#define LOG_BACK_FUNC  11
#define EXP_FUNC       12
#define INV_FUNC       13
#define INV_BACK_FUNC  14
#define IS_CLOSE_FUNC  15
#define MAX_FUNC       16
#define POW            17
#define TANH           18

__device__ float fn(int fn_id, float x, float y=0) {
    switch(fn_id) {
      case ADD_FUNC: {
        return x + y;
      }
      case MUL_FUNC: {
        return x * y;
      }
      case ID_FUNC: {
      	return x;
      }
      case NEG_FUNC: {
        return -x;
      }
      case LT_FUNC: {
        if (x < y) {
          return 1.0;
        }
        else {
          return 0.0;
        }
      }
      case EQ_FUNC: {
        if (x == y) {
          return 1.0;
        }
        else {
          return 0.0;
        }
      }
      case SIGMOID_FUNC: {
        if (x >= 0) {
          return 1.0 / (1.0 + exp(-x));
        }
        else {
          return exp(x) / (1.0 + exp(x));
        }
      }
      case RELU_FUNC: {
        return max(x, 0.0);
      }
      case RELU_BACK_FUNC: {
        if (x > 0) {
          return y;
        }
        else {
          return 0.0;
        }
      }
      case LOG_FUNC: {
        return log(x + 1e-6);
      }
      case LOG_BACK_FUNC: {
        return y / (x + 1e-6);
      }
      case EXP_FUNC: {
        return exp(x);
      }
      case INV_FUNC: {
        return float(1.0 / x);
      }
      case INV_BACK_FUNC: {
        return -(1.0 / (x * x)) * y;
      }
      case IS_CLOSE_FUNC: {
        return (x - y < 1e-2) && (y - x < 1e-2);
      }
      case MAX_FUNC: {
        if (x > y) {
          return x;
        }
        else {
          return y;
        }
      }
      case POW: {
	// TODO
        return powf(x, y);
      }
      case TANH: {
	// TODO
        return tanhf(x);
      }
      default: {
        return x + y;
      }
    }
    
}


__device__ int index_to_position(const int* index, const int* strides, int num_dims) {
    int position = 0;
    for (int i = 0; i < num_dims; ++i) {
        position += index[i] * strides[i];
    }
    return position;
}

__device__ void to_index(int ordinal, const int* shape, int* out_index, int num_dims) {
    int cur_ord = ordinal;
    for (int i = num_dims - 1; i >= 0; --i) {
        int sh = shape[i];
        out_index[i] = cur_ord % sh;
        cur_ord /= sh;
    }
}

__device__ void broadcast_index(const int* big_index, const int* big_shape, const int* shape, int* out_index, int num_dims_big, int num_dims) {
    for (int i = 0; i < num_dims; ++i) {
        if (shape[i] > 1) {
            out_index[i] = big_index[i + (num_dims_big - num_dims)];
        } else {
            out_index[i] = 0;
        }
    }
}


__global__ void MatrixMultiplyKernel(
    float* out,
    const int* out_shape,
    const int* out_strides,
    float* a_storage,
    const int* a_shape,
    const int* a_strides,
    float* b_storage,
    const int* b_shape,
    const int* b_strides
) {

    __shared__ float a_shared[TILE][TILE];
    __shared__ float b_shared[TILE][TILE];

    int bx = blockIdx.x, by = blockIdx.y;
    int tx = threadIdx.x, ty = threadIdx.y;

    int batch = blockIdx.z;
    int a_batch_stride = a_shape[0] > 1 ? a_strides[0] : 0;
    int b_batch_stride = b_shape[0] > 1 ? b_strides[0] : 0;

    int row = bx * blockDim.x + tx;
    int col = by * blockDim.y + ty;

    float pvalue = 0;

    for (int i = 0; i*TILE < a_shape[2]; ++i) {
        int a_col = i * TILE + ty;
        if(row < a_shape[1] && a_col < a_shape[2]) {
            a_shared[tx][ty] = a_storage[batch * a_batch_stride + row * a_strides[1] + a_col * a_strides[2]];
        } else {
            a_shared[tx][ty] = 0.0;
        }

        int b_row = i * TILE + tx;
        if(b_row < b_shape[1] && col < b_shape[2]) {
            b_shared[tx][ty] = b_storage[batch * b_batch_stride + b_row * b_strides[1] + col * b_strides[2]];
        } else {
            b_shared[tx][ty] = 0.0;
        }

        __syncthreads(); 

        for (int k = 0; k < TILE; ++k) {
          if ((i * TILE + k) < a_shape[2] && (i * TILE + k) < b_shape[1])
            pvalue += a_shared[tx][k] * b_shared[k][ty];
        }

        __syncthreads(); 
    }

    // 7. Write the computed value back to the global memory
    if (row < out_shape[1] && col < out_shape[2]) 
        out[batch * out_strides[0] + row * out_strides[1] + col * out_strides[2]] = pvalue;
}


__global__ void mapKernel(
    float* out, 
    int* out_shape, 
    int* out_strides, 
    int out_size, 
    float* in_storage, 
    int* in_shape, 
    int* in_strides,
    int shape_size,
    int fn_id
) {
    int out_index[MAX_DIMS];
    int in_index[MAX_DIMS];
    
    /// BEGIN ASSIGN1_2
    /// TODO
    // Hints:
    // 1. Compute the position in the output array that this thread will write to
    int position = blockIdx.x * blockDim.x + threadIdx.x;
    // additionally check if the position is valid
    if (position >= out_size) {
      return;
    }
    // 2. Convert the position to the out_index according to out_shape
    to_index(position, out_shape, out_index, shape_size);
    // 3. Broadcast the out_index to the in_index according to in_shape (optional in some cases)
    broadcast_index(out_index, out_shape, in_shape, in_index, shape_size, shape_size);
    // 4. Calculate the position of element in in_array according to in_index and in_strides
    int element_in = index_to_position(in_index, in_strides, shape_size);
    // 5. Calculate the position of element in out_array according to out_index and out_strides
    int element_out = index_to_position(out_index, out_strides, shape_size);
    // 6. Apply the unary function to the input element and write the output to the out memory
    // use the previously defined float type
    float input = in_storage[element_in];
    out[element_out] = fn(fn_id, input);
    
    /// END ASSIGN1_2
}


__global__ void reduceKernel(
    float* out, 
    int* out_shape, 
    int* out_strides, 
    int out_size, 
    float* a_storage, 
    int* a_shape, 
    int* a_strides, 
    int reduce_dim,
    float reduce_value,
    int shape_size,
    int fn_id
) {
    // __shared__ double cache[BLOCK_DIM]; // Uncomment this line if you want to use shared memory to store partial results
    int out_index[MAX_DIMS];

    /// BEGIN ASSIGN1_2
    /// TODO
    // 1. Define the position of the output element that this thread or this block will write to
    int position = blockIdx.x * blockDim.x + threadIdx.x;
    // additionally check if the position is valid
    if (position >= out_size) {
      return;
    }
    // 2. Convert the out_pos to the out_index according to out_shape
    to_index(position, out_shape, out_index, shape_size);
    // 3. Initialize the reduce_value to the output element
    float reduction_result = reduce_value;
    // 4. Iterate over the reduce_dim dimension of the input array to compute the reduced value
    for (int i = 0; i < a_shape[reduce_dim]; ++i) {
        // hold the indices for accessing elements in the input array
        int a_index[MAX_DIMS];
        // copy the out_index to a_index
        for (int j = 0; j < shape_size; ++j) {
            a_index[j] = out_index[j];
        }
        // set the current dimension for reduction
        a_index[reduce_dim] = i;
        int a_position = index_to_position(a_index, a_strides, shape_size);

        float input = a_storage[a_position];
        reduction_result = fn(fn_id, reduction_result, input);
    }
    // 5. Write the reduced value to out memory
    out[position] = reduction_result;
    /// END ASSIGN1_2
}

__global__ void zipKernel(
    float* out, 
    int* out_shape, 
    int* out_strides, 
    int out_size,
    int out_shape_size,
    float* a_storage, 
    int* a_shape, 
    int* a_strides,
    int a_shape_size,
    float* b_storage, 
    int* b_shape, 
    int* b_strides,
    int b_shape_size,
    int fn_id
) {
    int out_index[MAX_DIMS];
    int a_index[MAX_DIMS];
    int b_index[MAX_DIMS];

    /// BEGIN ASSIGN1_2
    /// TODO
    // Hints:
    // 1. Compute the position in the output array that this thread will write to
    int position = blockIdx.x * blockDim.x + threadIdx.x;
    // additionally check if the position is valid
    if (position >= out_size) {
      return;
    }
    // 2. Convert the position to the out_index according to out_shape
    to_index(position, out_shape, out_index, out_shape_size);
    // 3. Calculate the position of element in out_array according to out_index and out_strides
    int element_out = index_to_position(out_index, out_strides, out_shape_size);
    // 4. Broadcast the out_index to the a_index according to a_shape
    broadcast_index(out_index, out_shape, a_shape, a_index, out_shape_size, a_shape_size);
    // 5. Calculate the position of element in a_array according to a_index and a_strides
    int element_a = index_to_position(a_index, a_strides, a_shape_size);
    // 6. Broadcast the out_index to the b_index according to b_shape
    broadcast_index(out_index, out_shape, b_shape, b_index, out_shape_size, b_shape_size);
    // 7.Calculate the position of element in b_array according to b_index and b_strides
    int element_b = index_to_position(b_index, b_strides, b_shape_size);
    // 8. Apply the binary function to the input elements in a_array & b_array and write the output to the out memory
    // use the previously defined float type
    float input_a = a_storage[element_a];
    float input_b = b_storage[element_b];
    out[element_out] = fn(fn_id, input_a, input_b);

    /// END ASSIGN1_2
}


extern "C" {

void MatrixMultiply(
    float* out,
    int* out_shape,
    int* out_strides,
    float* a_storage,
    int* a_shape,
    int* a_strides,
    float* b_storage,
    int* b_shape,
    int* b_strides,
    int batch, int m, int p
) {
    int n = a_shape[2];

    // Allocate device memory
    float *d_out, *d_a, *d_b;
    hipMalloc(&d_a, batch * m * n * sizeof(float));
    hipMalloc(&d_b, batch * n * p * sizeof(float));
    hipMalloc(&d_out, batch * m * p * sizeof(float));

    int *d_out_shape, *d_out_strides, *d_a_shape, *d_a_strides, *d_b_shape, *d_b_strides;
    hipMalloc(&d_out_shape, 3 * sizeof(int));
    hipMalloc(&d_out_strides, 3 * sizeof(int));
    hipMalloc(&d_a_shape, 3 * sizeof(int));
    hipMalloc(&d_a_strides, 3 * sizeof(int));
    hipMalloc(&d_b_shape, 3 * sizeof(int));
    hipMalloc(&d_b_strides, 3 * sizeof(int));


    // Copy data to the device
    hipMemcpy(d_a, a_storage, batch * m * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b_storage, batch * n * p * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_out_shape, out_shape, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out_strides, out_strides, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_shape, a_shape, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_strides, a_strides, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_shape, b_shape, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_strides, b_strides, 3 * sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = BASE_THREAD_NUM;
    dim3 blockDims(threadsPerBlock, threadsPerBlock, 1); // Adjust these values based on your specific requirements
    dim3 gridDims((m + threadsPerBlock - 1) / threadsPerBlock, (p + threadsPerBlock - 1) / threadsPerBlock, batch);
    MatrixMultiplyKernel<<<gridDims, blockDims>>>(
        d_out, d_out_shape, d_out_strides, d_a, d_a_shape, d_a_strides, d_b, d_b_shape, d_b_strides
    );

    // Copy back to the host
    hipMemcpy(out, d_out, batch * m * p * sizeof(float), hipMemcpyDeviceToHost);
    
    hipDeviceSynchronize();

    // Check CUDA execution
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Matmul Error: %s\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    // Free memory on device
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
    hipFree(d_out_shape);
    hipFree(d_out_strides);
    hipFree(d_a_shape);
    hipFree(d_a_strides);
    hipFree(d_b_shape);
    hipFree(d_b_strides);
}

void tensorMap(
    float* out, 
    int* out_shape, 
    int* out_strides, 
    int out_size, 
    float* in_storage, 
    int* in_shape, 
    int* in_strides,
    int in_size,
    int shape_size,
    int fn_id
) {

    float *d_out, *d_in;
    hipMalloc(&d_out, out_size * sizeof(float));
    hipMalloc(&d_in, in_size * sizeof(float));

    int *d_out_shape, *d_out_strides, *d_in_shape, *d_in_strides;
    hipMalloc(&d_out_shape, shape_size * sizeof(int));
    hipMalloc(&d_out_strides, shape_size * sizeof(int));
    hipMalloc(&d_in_shape, shape_size * sizeof(int));
    hipMalloc(&d_in_strides, shape_size * sizeof(int));

    hipMemcpy(d_in, in_storage, in_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_out_shape, out_shape, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out_strides, out_strides, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_in_shape, in_shape, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_in_strides, in_strides, shape_size * sizeof(int), hipMemcpyHostToDevice);
    
    int threadsPerBlock = BASE_THREAD_NUM;
    int blocksPerGrid = (out_size + threadsPerBlock - 1) / threadsPerBlock;
    mapKernel<<<blocksPerGrid, threadsPerBlock>>>(
      d_out, d_out_shape, d_out_strides, out_size, 
      d_in, d_in_shape, d_in_strides, 
      shape_size, fn_id);
    
    // Copy back to the host
    hipMemcpy(out, d_out, out_size * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    // Check CUDA execution
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Map Error: %s\n", hipGetErrorString(err));
      // Handle the error (e.g., by exiting the program)
      exit(EXIT_FAILURE);
    }

    // Free memory on device
    hipFree(d_in);
    hipFree(d_out);
    hipFree(d_out_shape);
    hipFree(d_out_strides);
    hipFree(d_in_shape);
    hipFree(d_in_strides);
}


void tensorZip(
    float* out, 
    int* out_shape, 
    int* out_strides, 
    int out_size,
    int out_shape_size,
    float* a_storage, 
    int* a_shape, 
    int* a_strides,
    int a_size,
    int a_shape_size,
    float* b_storage, 
    int* b_shape, 
    int* b_strides,
    int b_size,
    int b_shape_size,
    int fn_id
) {

    // Allocate device memory
    float *d_out, *d_a, *d_b;
    hipMalloc((void **)&d_a, a_size * sizeof(float));
    hipMalloc(&d_b, b_size * sizeof(float));
    hipMalloc(&d_out, out_size * sizeof(float));

    int *d_out_shape, *d_out_strides, *d_a_shape, *d_a_strides, *d_b_shape, *d_b_strides;
    hipMalloc(&d_out_shape, out_shape_size * sizeof(int));
    hipMalloc(&d_out_strides, out_shape_size * sizeof(int));
    hipMalloc(&d_a_shape, a_shape_size * sizeof(int));
    hipMalloc(&d_a_strides, a_shape_size * sizeof(int));
    hipMalloc(&d_b_shape, b_shape_size * sizeof(int));
    hipMalloc(&d_b_strides, b_shape_size * sizeof(int));

    // Copy data to the device
    hipMemcpy(d_a, a_storage, a_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b_storage, b_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_out_shape, out_shape, out_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out_strides, out_strides, out_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_shape, a_shape, a_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_strides, a_strides, a_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_shape, b_shape, b_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_strides, b_strides, b_shape_size * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = BASE_THREAD_NUM;
    int blocksPerGrid = (out_size + threadsPerBlock - 1) / threadsPerBlock;
    zipKernel<<<blocksPerGrid, threadsPerBlock>>>(
      d_out, d_out_shape, d_out_strides, out_size, out_shape_size,
      d_a, d_a_shape, d_a_strides, a_shape_size,
      d_b, d_b_shape, d_b_strides, b_shape_size,
      fn_id);

    // Copy back to the host
    hipMemcpy(out, d_out, out_size * sizeof(float), hipMemcpyDeviceToHost);
    
    hipDeviceSynchronize();


    // Check CUDA execution
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Zip Error: %s\n", hipGetErrorString(err));
      // Handle the error (e.g., by exiting the program)
      exit(EXIT_FAILURE);
    }

    // Free memory on device
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
    hipFree(d_out_shape);
    hipFree(d_out_strides);
    hipFree(d_a_shape);
    hipFree(d_a_strides);
    hipFree(d_b_shape);
    hipFree(d_b_strides);
}



void tensorReduce(
    float* out, 
    int* out_shape, 
    int* out_strides, 
    int out_size, 
    float* a_storage, 
    int* a_shape, 
    int* a_strides, 
    int reduce_dim, 
    float reduce_value,
    int shape_size,
    int fn_id
) {
    int a_size = out_size * a_shape[reduce_dim];
    float *d_out, *d_a;
    hipMalloc(&d_out, out_size * sizeof(float));
    hipMalloc(&d_a, a_size * sizeof(float));

    int *d_out_shape, *d_out_strides, *d_a_shape, *d_a_strides;
    hipMalloc(&d_out_shape, shape_size * sizeof(int));
    hipMalloc(&d_out_strides, shape_size * sizeof(int));
    hipMalloc(&d_a_shape, shape_size * sizeof(int));
    hipMalloc(&d_a_strides, shape_size * sizeof(int));

    hipMemcpy(d_a, a_storage, a_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_out_shape, out_shape, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out_strides, out_strides, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_shape, a_shape, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_strides, a_strides, shape_size * sizeof(int), hipMemcpyHostToDevice);
    
    int threadsPerBlock = BASE_THREAD_NUM;
    int blocksPerGrid = (out_size + threadsPerBlock - 1) / threadsPerBlock;
    reduceKernel<<<blocksPerGrid, threadsPerBlock>>>(
        d_out, d_out_shape, d_out_strides, out_size, 
        d_a, d_a_shape, d_a_strides, 
        reduce_dim, reduce_value, shape_size, fn_id
    );

    // Copy back to the host
    hipMemcpy(out, d_out, out_size * sizeof(float), hipMemcpyDeviceToHost);
    
    hipDeviceSynchronize();

    // Check CUDA execution
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Reduce Error: %s\n", hipGetErrorString(err));
      // Handle the error (e.g., by exiting the program)
      exit(EXIT_FAILURE);
    }

    hipFree(d_a);
    hipFree(d_out);
    hipFree(d_out_shape);
    hipFree(d_out_strides);
    hipFree(d_a_shape);
    hipFree(d_a_strides);
}

}